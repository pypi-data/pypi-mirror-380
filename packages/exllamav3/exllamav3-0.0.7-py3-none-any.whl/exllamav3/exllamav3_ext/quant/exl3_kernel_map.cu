#include <hip/hip_fp16.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;
#include "../util.h"
#include "../util.cuh"
#include "../ptx.cuh"
#include <tuple>
#include <mutex>
#include "exl3_kernel_map.cuh"
#include "exl3_devctx.cuh"
#include "comp_units/exl3_comp_unit_1.cuh"
#include "comp_units/exl3_comp_unit_2.cuh"
#include "comp_units/exl3_comp_unit_3.cuh"
#include "comp_units/exl3_comp_unit_4.cuh"
#include "comp_units/exl3_comp_unit_5.cuh"
#include "comp_units/exl3_comp_unit_6.cuh"
#include "comp_units/exl3_comp_unit_7.cuh"
#include "comp_units/exl3_comp_unit_8.cuh"

int select_gemm_shape(int cc, int size_m, int size_k, int size_n, int bits, bool multi, int bszm_in, int bszm_out)
{
    bool mod_256 = (size_n % 256 == 0);
    bool mod_512 = (size_n % 512 == 0);

    size_k *= bszm_in;
    size_n *= bszm_out;

    switch(cc)
    {
        case CC_OLD:
        case CC_AMPERE:
            if (mod_256 && bits <= 4)
            {
                if (size_n <= 2048 || size_k <= 2048) return 2;
                return 3;
            }
            if (mod_256 && size_n < 4096) return size_k > 8192 ? 3 : 2;
            if (mod_512 && (size_n * size_k) > (4096 * 4096) && bits <= 6) return 4;
            if (mod_256) return 3;
            return 2;

        case CC_ADA:
            if (mod_256 && bits <= 3)
            {
                if (size_k <= 2048 && !multi) return 2;
                if (size_n < 4096 && size_k <= 12288) return 2;
                return 3;
            }
            if (size_n <= 16384) return 2;
            if (mod_512 && size_n >= 32768) return 4;
            if (mod_256) return 3;
            return 2;

        case CC_HOPPER:
        case CC_BLACKWELL:
            if ((bits == 4 || bits == 2) && !multi)
            {
                if (size_k <= 2048) return 1;
            }
            if (bits >= 7)
            {
                if (mod_256 && size_n <= 8192) return size_k > 32768 ? 3 : 2;
                if (mod_512 && size_n > 32768) return 4;
                return 2;
            }
            if (mod_256 && size_n <= 4096) return size_k > 8192 && bits >= 3 ? 3 : 2;
            if (mod_512 && size_n > 16384) return 4;
            if (mod_256) return 3;
            return 2;
    }
    return 0;
}

int exl3_gemm_num_kernel_shapes()
{
    return EXL3_GEMM_NUM_SHAPES;
}

int exl3_gemm_tilesize_k[] = {EXL3_GEMM_TILESIZE_K};
int exl3_gemm_tilesize_n[] = {EXL3_GEMM_TILESIZE_N};
int exl3_gemm_blockdim[] = {EXL3_GEMM_BLOCKDIM};

bool exl3_gemm_shape_compat(int shape_idx, int size_m, int size_k, int size_n, int bits)
{
    int tilesize_k = exl3_gemm_tilesize_k[shape_idx];
    int tilesize_n = exl3_gemm_tilesize_n[shape_idx];
    return (size_k % tilesize_k == 0) && (size_n % tilesize_n == 0);
}

fp_exl3_gemm_kernel select_exl3_gemm_kernel
(
    int cc,
    int size_m,
    int size_k,
    int size_n,
    int bits,
    bool c_fp32,
    int force_shape_idx,
    int* out_block_dim,
    int* out_shape_idx,
    int* num_sms,
    int cb
)
{
    int shape_idx = force_shape_idx <= 0 ? select_gemm_shape(cc, size_m, size_k, size_n, bits, false, 1, 1) : force_shape_idx;

    TORCH_CHECK(shape_idx > 0, "exl3_gemm: no compatible kernel");
    if (out_shape_idx) *out_shape_idx = shape_idx;
    if (out_block_dim) *out_block_dim = exl3_gemm_blockdim[shape_idx];

    // Avoid empty blocks
    if (num_sms)
    {
        int tilesize_k = exl3_gemm_tilesize_k[shape_idx];
        int tilesize_n = exl3_gemm_tilesize_n[shape_idx];
        int max_slices = size_k / tilesize_k * size_n / tilesize_n;
        *num_sms = MAX(MIN(max_slices, *num_sms), 1);
    }

    int kernel_idx = shape_idx + (EXL3_GEMM_NUM_SHAPES + 1) * cb;

    if (c_fp32)
    {
        switch (bits)
        {
            case 1: return tfp_exl3_gemm_kernel_fp32_b1[kernel_idx];
            case 2: return tfp_exl3_gemm_kernel_fp32_b2[kernel_idx];
            case 3: return tfp_exl3_gemm_kernel_fp32_b3[kernel_idx];
            case 4: return tfp_exl3_gemm_kernel_fp32_b4[kernel_idx];
            case 5: return tfp_exl3_gemm_kernel_fp32_b5[kernel_idx];
            case 6: return tfp_exl3_gemm_kernel_fp32_b6[kernel_idx];
            case 7: return tfp_exl3_gemm_kernel_fp32_b7[kernel_idx];
            case 8: return tfp_exl3_gemm_kernel_fp32_b8[kernel_idx];
            default: TORCH_CHECK(false, "No kernel for GEMM shape");
        }
    }
    else
    {
        switch (bits)
        {
            case 1: return tfp_exl3_gemm_kernel_fp16_b1[kernel_idx];
            case 2: return tfp_exl3_gemm_kernel_fp16_b2[kernel_idx];
            case 3: return tfp_exl3_gemm_kernel_fp16_b3[kernel_idx];
            case 4: return tfp_exl3_gemm_kernel_fp16_b4[kernel_idx];
            case 5: return tfp_exl3_gemm_kernel_fp16_b5[kernel_idx];
            case 6: return tfp_exl3_gemm_kernel_fp16_b6[kernel_idx];
            case 7: return tfp_exl3_gemm_kernel_fp16_b7[kernel_idx];
            case 8: return tfp_exl3_gemm_kernel_fp16_b8[kernel_idx];
            default: TORCH_CHECK(false, "No kernel for GEMM shape");
        }
    }
}

fp_exl3_mgemm_kernel select_exl3_mgemm_kernel
(
    int cc,
    int size_m,
    int size_k,
    int size_n,
    int bits,
    bool c_fp32,
    int force_shape_idx,
    int* out_block_dim,
    int* out_shape_idx,
    int* num_sms,
    int cb,
    int bszm_in,
    int bszm_out
)
{
    int shape_idx = force_shape_idx <= 0 ? select_gemm_shape(cc, size_m, size_k, size_n, bits, true, bszm_in, bszm_out) : force_shape_idx;
    TORCH_CHECK(shape_idx > 0, "exl3_mgemm: no compatible kernel");
    if (out_shape_idx) *out_shape_idx = shape_idx;
    if (out_block_dim) *out_block_dim = exl3_gemm_blockdim[shape_idx];

    // Avoid empty blocks
    if (num_sms)
    {
        int tilesize_k = exl3_gemm_tilesize_k[shape_idx];
        int tilesize_n = exl3_gemm_tilesize_n[shape_idx];
        int max_slices = size_k / tilesize_k * size_n / tilesize_n / (*num_sms > 128 ? 20 : 24);
        *num_sms = MIN(max_slices, *num_sms);
    }

    int kernel_idx = shape_idx + (EXL3_GEMM_NUM_SHAPES + 1) * cb;

    if (c_fp32)
    {
        switch (bits)
        {
            case 1: return tfp_exl3_mgemm_kernel_fp32_b1[kernel_idx];
            case 2: return tfp_exl3_mgemm_kernel_fp32_b2[kernel_idx];
            case 3: return tfp_exl3_mgemm_kernel_fp32_b3[kernel_idx];
            case 4: return tfp_exl3_mgemm_kernel_fp32_b4[kernel_idx];
            case 5: return tfp_exl3_mgemm_kernel_fp32_b5[kernel_idx];
            case 6: return tfp_exl3_mgemm_kernel_fp32_b6[kernel_idx];
            case 7: return tfp_exl3_mgemm_kernel_fp32_b7[kernel_idx];
            case 8: return tfp_exl3_mgemm_kernel_fp32_b8[kernel_idx];
            default: TORCH_CHECK(false, "No kernel for GEMM shape");
        }
    }
    else
    {
        switch (bits)
        {
            case 1: return tfp_exl3_mgemm_kernel_fp16_b1[kernel_idx];
            case 2: return tfp_exl3_mgemm_kernel_fp16_b2[kernel_idx];
            case 3: return tfp_exl3_mgemm_kernel_fp16_b3[kernel_idx];
            case 4: return tfp_exl3_mgemm_kernel_fp16_b4[kernel_idx];
            case 5: return tfp_exl3_mgemm_kernel_fp16_b5[kernel_idx];
            case 6: return tfp_exl3_mgemm_kernel_fp16_b6[kernel_idx];
            case 7: return tfp_exl3_mgemm_kernel_fp16_b7[kernel_idx];
            case 8: return tfp_exl3_mgemm_kernel_fp16_b8[kernel_idx];
            default: TORCH_CHECK(false, "No kernel for GEMM shape");
        }
    }
}
#include <stdio.h>

#include "async_stream.cuh"

bool g_AsyncStreamInitialized = false;
hipStream_t g_AsyncStream;

int AsyncInitStreamIfNeeded() {
    if (!g_AsyncStreamInitialized) {
        const hipError_t err = hipStreamCreate(&g_AsyncStream);

        if (err != hipSuccess) {
            fprintf(stderr, "Error creating CUDA stream: %s\n", hipGetErrorString(err));
            return -1;
        }

        hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
        g_AsyncStreamInitialized = true;
    }

    return 0;
}

hipStream_t* GetAsyncStream() {
    return &g_AsyncStream;
}

int AsyncDestroyStreamIfNeeded() {
    if (g_AsyncStreamInitialized) {
        if (const hipError_t err = hipStreamDestroy(g_AsyncStream); err != hipSuccess) {
            fprintf(stderr, "Error destroying CUDA stream: %s\n", hipGetErrorString(err));
            return -1;
        }

        g_AsyncStreamInitialized = false;
    }

    return 0;
}

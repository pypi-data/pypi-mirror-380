#include "hip/hip_runtime.h"
// Hadamard gate CUDA kernel
__global__ void hadamard_kernel(hipFloatComplex* state, int qubit, int n_states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_states) return;
    
    int mask = 1 << qubit;
    if ((idx & mask) == 0) {
        int partner = idx | mask;
        if (partner < n_states) {
            hipFloatComplex a = state[idx];
            hipFloatComplex b = state[partner];
            
            // H|0⟩ = (|0⟩ + |1⟩)/√2, H|1⟩ = (|0⟩ - |1⟩)/√2
            float sqrt2_inv = 0.7071067811865475f; // 1/√2
            state[idx] = make_hipFloatComplex(
                sqrt2_inv * (hipCrealf(a) + hipCrealf(b)),
                sqrt2_inv * (hipCimagf(a) + hipCimagf(b))
            );
            state[partner] = make_hipFloatComplex(
                sqrt2_inv * (hipCrealf(a) - hipCrealf(b)),
                sqrt2_inv * (hipCimagf(a) - hipCimagf(b))
            );
        }
    }
}